#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

__global__ void add( int a , int b, int* sum )
{
	*sum = a + b;
}

int add(int a, int b)
{
	int hostSum = 0;
	int* cudaSum = NULL;
	// Allocate Space
	CUDA_CHECK( hipMalloc((void**)&cudaSum, sizeof(int)) );

	add<<<1, 1>>>( a, b, cudaSum );
	CUDA_CHECK( hipMemcpy( &hostSum, cudaSum, sizeof(int), hipMemcpyDeviceToHost ) );

	// Free Space
	CUDA_CHECK( hipFree( cudaSum ) );

	return hostSum;
}
