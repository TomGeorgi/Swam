#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

__global__ void add( int a , int b, int* sum )
{
	*sum = a + b;
}

__global__ void multiplyVM(double* matrix, double* vector, int rows, int cols, double* result)
{
	printf("rows %i, cols %i\n", rows, cols);
	for (int c = 0; c < cols; c++) printf("%d\n", vector[c]);
	for (int r = 0; r < rows; r++)
	{
		double tmp = 0;
		for (int c = 0; c < cols; c++)
		{
			double m_val = matrix[(r * cols) + c];
			double v_val = vector[c];
			printf("m_val = %d\n", m_val);
			printf("v_val = %d\n", v_val);
			printf("r = %i, c = %i\n", r, c);
			tmp += m_val * v_val;
			printf("tmp = %d\n", tmp);
		}
		result[r] = tmp;
	}

	printf("multiplying finis\n");
}

int add(int a, int b)
{
	int hostSum = 0;
	int* cudaSum = NULL;
	// Allocate Space
	CUDA_CHECK( hipMalloc((void**)&cudaSum, sizeof(int)) );

	add<<<1, 1>>>( a, b, cudaSum );
	CUDA_CHECK( hipMemcpy( &hostSum, cudaSum, sizeof(int), hipMemcpyDeviceToHost ) );

	// Free Space
	CUDA_CHECK( hipFree( cudaSum ) );

	return hostSum;
}

void multiply()
{
	const int rows = 2;
	const int cols = 3;

	// Declare on host
	double** A = new double* [rows];
	A[0] = new double[rows * cols]{2, 5, 3, 3, 4, 2};
	for (int i = 1; i < rows; ++i) A[i] = A[i - 1] + cols;

	double hostVector[cols] = { 2, 3, 1 };
	double hostResult[cols] = { 0, 0 };

	// Declare for CUDA
	double* cudaMatrix = NULL;
	double* cudaVector = NULL;
	double* cudaResult = NULL;

	// Allocate Space
	std::cout << "allocate memory" << std::endl;
	CUDA_CHECK(hipMalloc((void**)&cudaMatrix, sizeof(double) * rows * cols));
	CUDA_CHECK(hipMalloc((void**)&cudaVector, sizeof(double) * cols));
	CUDA_CHECK(hipMalloc((void**)&cudaResult, sizeof(double) * rows));

	std::cout << "memcpy to dev" << std::endl;
	CUDA_CHECK(hipMemcpy(cudaMatrix, A[0], sizeof(double) * rows * cols, hipMemcpyHostToDevice));
	std::cout << "memcpy to dev" << std::endl;
	CUDA_CHECK(hipMemcpy(cudaVector, hostVector, sizeof(double) * cols, hipMemcpyHostToDevice));

	multiplyVM<<<1, 1>>>(cudaMatrix, cudaVector, rows, cols, cudaResult);
	CUDA_CHECK(hipMemcpy(hostResult, cudaResult, sizeof(double) * rows, hipMemcpyDeviceToHost));

	cout << "result:" << endl; // [20, 20]
	for (int i = 0; i < rows; i++)
	{
		cout << hostResult[i] << endl;
	}
	CUDA_CHECK(hipFree(cudaVector));
	CUDA_CHECK(hipFree(cudaMatrix));
	CUDA_CHECK(hipFree(cudaResult));
}
